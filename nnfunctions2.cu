
#include <hip/hip_runtime.h>
// #include "nnfunctions.h"

// #ifndef SIZE
// #define SIZE 32
// #endif

// __device__ void matMult(int N, int M, int P, float *A, float *B, float *C) {

//   __shared__ float sA[SIZE][SIZE];
//   __shared__ float sB[SIZE][SIZE];

//   int bx = blockIdx.x;  int by = blockIdx.y;
//   int tx = threadIdx.x; int ty = threadIdx.y;
//   int row = by * SIZE + ty;
//   int col = bx * SIZE + tx;
//   int k, m;

//   float tmp = 0.0;
//   for (m=0; m < P-SIZE; m=m+SIZE) {
//     if (row<N) sA[ty][tx] = A[row*P + m + tx];
//     if (col<M) sB[ty][tx] = B[col + (m + ty)*M];
//     __syncthreads();

//     for (k=0; k<SIZE; k++)
//       tmp += sA[ty][k] * sB[k][tx];

//     __syncthreads();
//   }
//   if (row<N) sA[ty][tx] = A[row*P + m + tx];
//   if (col<M) sB[ty][tx] = B[col + (m + ty)*M];
//   __syncthreads();
//   for (k=0; m<P; k++, m++)
//     tmp += sA[ty][k] * sB[k][tx];

//   if (row<N && col<M) C[row*M+col] = tmp;

// }

// __global__ void globalMatMult(int N, int M, int P, float *A, float *B, float *C) {
//     matMult(N,M,P,A,B,C);
// }


// __device__ void sigmoid(int N, float *input, float *output) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < N) {
//         output[i] = 1 / (1 + exp(-1*input[i]));
//     }
// }

// __global__ void globalSigmoid(int N, float *input, float *output) {
//   sigmoid(N,input, output);
// }

// __device__ void reLU(int N, float *input, float *output) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < N) {
//         if (input[i] < 0) output[i] = 0;
//         else output[i] = input[i];
//     }
// }

// __global__ void globalReLU(int N, float *input, float *output) {
//   reLU(N,input,output);
// }

// __global__ void costFunction(int N, int nFeatures, float *Z, float *Y, float *odata){

//   __shared__ float tmpd[N];
//   int tid = threadIdx.x;
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;

//   tmpd[tid] = (Y[idx] * logf(Z[idx])) + ((1 - Y[idx]) * logf(1 - Z[idx]));

//   // Synchronize threads within the block
//   __syncthreads();

//   // Hacemos la reduccion en la memoria compartida
//   for (int s=blockDim.x/2; s>32; s>>=1) {
//     if (tid < s)
//       sdata[tid] += sdata[tid + s];
//     __syncthreads();
//   }
//   // desenrrollamos el ultimo warp activo
//   if (tid < 32) {
//     volatile double *smem = sdata;

//     smem[tid] += smem[tid + 32];
//     smem[tid] += smem[tid + 16];
//     smem[tid] += smem[tid + 8];
//     smem[tid] += smem[tid + 4];
//     smem[tid] += smem[tid + 2];
//     smem[tid] += smem[tid + 1];
//   }


//   // El thread 0 escribe el resultado de este bloque en la memoria global
//   if (tid == 0) odata[blockIdx.x] = sdata[0]/nFeatures;
// }

// __global__ void dotProd(int N, float *vec1, float *vec2, float *res){
//   __shared__ float tmpd[N];
//   int tid = threadIdx.x;
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;

//   // Perform the dot product calculation
//   tmpd[tid] = vec1[idx] * vec2[idx];

//   // Synchronize threads within the block
//   __syncthreads();

//   // Perform parallel reduction
//   for (int s=blockDim.x/2; s>32; s>>=1) {
//     if (tid < s)
//       tmpd[tid] += tmpd[tid + s];
//     __syncthreads();
//   }
//   // desenrrollamos el ultimo warp activo
//   if (tid < 32) {
//     volatile double *smem = tmpd;

//     smem[tid] += smem[tid + 32];
//     smem[tid] += smem[tid + 16];
//     smem[tid] += smem[tid + 8];
//     smem[tid] += smem[tid + 4];
//     smem[tid] += smem[tid + 2];
//     smem[tid] += smem[tid + 1];
//   }
//   // El thread 0 escribe el resultado de este bloque en la memoria global
//   if (tid == 0) res[blkId] = tmpd[0];
// }

// __device__ void transposeMatrix(float *A, float *B, int row, int col){


//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   int idy = blockIdx.y * blockDim.y + threadIdx.y;

//   if(idx < col && idy < row){
//     B[idy + row*idx] = A[idx + col*idy];
//   }
// }
// __global__ void updateLayers( double alpha, 
//                               float* dW1, float *dW2
//                               float *W1, float *W2){
//     /*
//     Updates weight, biases suing gradient descendent method using backprop
//     Given:
//       -derivative w
//       -derivative b
//       -alpha learning rate (own set)
//     Output:
//       Updates current layer's weights and biases 
//     */

//     //for each layer:
//     //W_i = W_i - aplha * dW_i
//     //b_i = b_i - alpha * db1

//     scalarProdMat<<<grid, block>>>(nOutput, nOutput, alpha, dW1, dW1alpha);
//     substractMat<<<grid, block>>>(nHiddenLayer, nOutput, W1, dW1alpha);

//     scalarProdMat<<<grid, block>>>(nFeatures, nHiddenLayer, alpha, dW2, dW2alpha);
//     substractMat<<<grid, block>>>(nFeatures, nHiddenLayer, W2, dW2alpha);

// }
// __global__ void backprop(int nFeatures, int batchSize, int nHiddenLayer, int nOutput, int nLayers,
//                         float *X, float *hiddenWeights, float *outputWeights, float *actL1, float *actL2, float *Y,
//                         float *dZ1, float dZ2, float dW1, float *dW2, float *db1, float *db2) {
//     /*
//     Given:
//       - number of input data m
//       - z that is z = w*a + b
//       - w weight vector
//       - b biases vector
    
//     Output
//       - derivative z respective to error
//       - derivative w respective to error
//       - derivative b respective to error
//     */

//     // the following lanes must be done trhough every layer
//     //dZ[last_column] 2 = A[last_column] - Y
//     //dW[last_column] 2= 1 / m * dZ[last_column] * transpose(A[last_column - 1])
//     //db[last_column] 2= 1 / m * sum(dZ[last_column])
    
//     //dZ[last_column - 1] 1= trasnpose(W[last_column]) *(dot product) dZ[last_column] * derivative(Z[last_column-1])
//     //dW[last_column - 1] 1= 1 / m * dZ1 *(dot prod) transpose(X)
//     //db[last_column - 1] 1= 1 / m * sum(dZ[last_column - 1])

//     /*__shared__ float tmpdZ2[nOutput]

//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if(idx < batchSize){
      
//       int l2 = nLayers - 1;
//       //Derivative Z output layer Out[2]
//       for(int i = 0; i < nOutput; ++i){
//         dZ2[idx*nOutput + i] = actL2[idx*nOutput + i] - Y[idx*nOutput + i];
//       }
//       //Derivative W
//       for(int i = 0; i < nOutput; ++i){
//         //dW[idx*nOutput + i] = 1/nFeatures * dZ[idx + nOutput*layer2]
//         transpose(actL1, res);
//         matMult(N, M, P, dZ, res, c);
//         dW[idx*nOutput + i] = c;
//       }
//       //Derivative b
      
//     }*/ç
//     // Derivative dZ2
//     substractMat<<<6, 10>>>(nOutput, batchSize, actL2, Y, dZ2);
//     // Derivative dW2
//     transpose<<<6, 10>>>(nOutput, batchSize, actL1, actL1T);
//     matMult<<<grid, block>>>(nOutput, nOutput, batchSize, dZ2, actL1T, res);
//     scalarDivMat<<<grid, block>>>(nOutput, nOutput, batchSize, res, dW2);
//     // Derivative db2


//     // Derivative Z1
//     derivativeReLu<<<grid, block>>>(nOutput, batchSize, Z1, gZ1);
//     transpose<<<6, 10>>>(nHiddenLayer, nOutput, w2, w2T);
//     matMult<<<grid, block>>>(nOutput, batchSize,nOutput, w2T, dZ2, aux2);
//     elementWiseProd<<<grid, block>>>(nOutput, batchSize, aux2, gZ1, dZ1);
    
//     //Derivative W1
//     transpose<<<6, 10>>>(batchSize, nFeatures, X, XT);
//     matMult<<<grid, block>>>(nOutput, nFeatures, batchSize, dZ1, XT, dW1aux);
//     scalarDivMat<<<grid, block>>>(nOutput, nFeatures, batchSize, dW1aux, dW1);
// }

// __global__ void elementWiseProd(int N, int M, float *A, float *B, float *C) {
//     int i = blockIdx.y * blockDim.y + threadIdx.y;
//     int j = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < N && j < M) {
//         C[i * M + j] = A[i * M + j] * B[i * M + j];
//     }
// }

// __global__ void subtractMat(int N, int M, float *A, float *B, float *C) {
//     int i = blockIdx.y * blockDim.y + threadIdx.y;
//     int j = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < N && j < M) {
//         C[i * M + j] = A[i * M + j] - B[i * M + j];
//     }
// }

// __global__ void scalarDivMat(int N, int M, float value, float *A, float *C) {
//     int i = blockIdx.y * blockDim.y + threadIdx.y;
//     int j = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < N && j < M) {
//         C[i * M + j] = A[i * M + j] / value;
//     }
// }

// __global__ void scalarProdMat(int N, int M, float value, float *A, float *C) {
//     int i = blockIdx.y * blockDim.y + threadIdx.y;
//     int j = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < N && j < M) {
//         C[i * M + j] = A[i * M + j] * value;
//     }
// }

// __global__ int derivative(int N, int M, float *A, float *C){
//     int i = blockIdx.y * blockDim.y + threadIdx.y;
//     int j = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < N && j < M) {
//         if (A[i*M + j] > 0) C[i*M + j] = 1;
//         else C[i*M + j] = 0;
//     }
    
// }

// __global__ void forwardPass(int nFeatures, int batchSize, int nHiddenLayer, int nOutput,
//                             float *input, float *hiddenWeights, float *activationL1, float *outputWeights, float *result) {

//   // matMult(batchSize, nHiddenLayer, nFeatures, input, hiddenWeights, activationL1);
//   // sigmoid(batchSize * nHiddenLayer, activationL1);
//   // matMult(batchSize, nOutput, nHiddenLayer, activationL1, outputWeights, result);
//   // sigmoid(batchSize * nHiddenLayer, result);

// }

