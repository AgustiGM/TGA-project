#include "hip/hip_runtime.h"
#include "nnfunctions.h"
#include <stdio.h>

#ifndef SIZE
#define SIZE 32
#endif

// struct Layer {
//     int nInput;
//     int nOutput;
//     float *weights;
//     float *biases;
//     float *activations;
// };

// C(N × M) ← A(N × P) · B (P × M)
/**__global__ void matMult(int N, int M, int P, float *A, float *B, float *C)
{

  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;
  int k, m;

  float tmp = 0.0;
  for (m = 0; m < P - SIZE; m = m + SIZE)
  {
    if (row < N)
      sA[ty][tx] = A[row * P + m + tx];
    if (col < M)
      sB[ty][tx] = B[col + (m + ty) * M];
    __syncthreads();

    for (k = 0; k < SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];

    __syncthreads();
  }
  if (row < N)
    sA[ty][tx] = A[row * P + m + tx];
  if (col < M)
    sB[ty][tx] = B[col + (m + ty) * M];
  __syncthreads();
  for (k = 0; m < P; k++, m++)
    tmp += sA[ty][k] * sB[k][tx];

  if (row < N && col < M)
    C[row * M + col] = tmp;
}**/

__global__ void sigmoid(int N, float *input, float *output)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = tid; i < N; i += blockDim.x * gridDim.x)
  {
    output[i] = 1.0f / (1.0f + exp(-1.0f * input[i]));
  }
}

__device__ void reLU(int N, float *input, float *output)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = tid; i < N; i += blockDim.x * gridDim.x)
  {
    if (input[i] > 0)
    {
      output[i] = input[i];
    }
    else
    {
      output[i] = 0;
    }
  }
}

__global__ void globalReLU(int N, float *input, float *output)
{
  reLU(N, input, output);
}

<<<<<<< HEAD
__global__ void costFunction(int N, int nFeatures, float *Z, float *Y, float *odata){

  __shared__ float tmpd[N];
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  tmpd[tid] = (Y[idx] * logf(Z[idx])) + ((1 - Y[idx]) * logf(1 - Z[idx]));

  // Synchronize threads within the block
  __syncthreads();

  // Hacemos la reduccion en la memoria compartida
  for (int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  // desenrrollamos el ultimo warp activo
  if (tid < 32) {
    volatile double *smem = sdata;

    smem[tid] += smem[tid + 32];
    smem[tid] += smem[tid + 16];
    smem[tid] += smem[tid + 8];
    smem[tid] += smem[tid + 4];
    smem[tid] += smem[tid + 2];
    smem[tid] += smem[tid + 1];
  }


  // El thread 0 escribe el resultado de este bloque en la memoria global
  if (tid == 0) odata[blockIdx.x] = sdata[0]/nFeatures;
}

__global__ void dotProd(int N, float *vec1, float *vec2, float *res){
  __shared__ float tmpd[N];
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Perform the dot product calculation
  tmpd[tid] = vec1[idx] * vec2[idx];

  // Synchronize threads within the block
  __syncthreads();

  // Perform parallel reduction
  for (int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s)
      tmpd[tid] += tmpd[tid + s];
    __syncthreads();
  }
  // desenrrollamos el ultimo warp activo
  if (tid < 32) {
    volatile double *smem = tmpd;

    smem[tid] += smem[tid + 32];
    smem[tid] += smem[tid + 16];
    smem[tid] += smem[tid + 8];
    smem[tid] += smem[tid + 4];
    smem[tid] += smem[tid + 2];
    smem[tid] += smem[tid + 1];
  }
  // El thread 0 escribe el resultado de este bloque en la memoria global
  if (tid == 0) res[blkId] = tmpd[0];
}
__global__ int derivative(float Z){
    //if (Z > 0) return 1; 
    //else return 0;
}

__device__ void transposeMatrix(float *A, float *B, int row, int col){


  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  if(idx < col && idy < row){
    B[idy + row*idx] = A[idx + col*idy];
  }
}
__global__ void updateLayers(){
    /*
    Updates weight, biases suing gradient descendent method using backprop
    Given:
      -derivative w
      -derivative b
      -alpha learning rate (own set)
    Output:
      Updates current layer's weights and biases 
    */

    //for each layer:
    //W_i = W_i - aplha * dW_i
    //b_i = b_i - alpha * db1
}
__global__ void backprop(int nFeatures, int batchSize, int nHiddenLayer, int nOutput, int nLayers,
                        float *hiddenWeights, float *outputWeights, float *actL1, float *actL2, float *Y,
                        float *dZ1, float dZ2, float dW1, float *dW2, float *db1, float *db2) {
    /*
    Given:
      - number of input data m
      - z that is z = w*a + b
      - w weight vector
      - b biases vector
    
    Output
      - derivative z respective to error
      - derivative w respective to error
      - derivative b respective to error
    */

    // the following lanes must be done trhough every layer
    //dZ[last_column] 2 = A[last_column] - Y
    //dW[last_column] 2= 1 / m * dZ[last_column] * transpose(A[last_column - 1])
    //db[last_column] 2= 1 / m * sum(dZ[last_column])
    
    //dZ[last_column - 1] 1= trasnpose(W[last_column]) *(dot product) dZ[last_column] * derivative(Z[last_column-1])
    //dW[last_column - 1] 1= 1 / m * dZ1 *(dot prod) transpose(X)
    //db[last_column - 1] 1= 1 / m * sum(dZ[last_column - 1])

    __shared__ float tmpdZ2[nOutput]

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < batchSize){
      
      int l2 = nLayers - 1;
      //Derivative Z output layer Out[2]
      for(int i = 0; i < nOutput; ++i){
        dZ2[idx*nOutput + i] = actL2[idx*nOutput + i] - Y[idx*nOutput + i];
      }
      //Derivative W
      for(int i = 0; i < nOutput; ++i){
        //dW[idx*nOutput + i] = 1/nFeatures * dZ[idx + nOutput*layer2]
        transpose(actL1, res);
        matMult(N, M, P, dZ, res, c);
        dW[idx*nOutput + i] = c;
      }
      //Derivative b
      
    }
=======
__global__ void backprop(int N, float *A)
{
}

__device__ float localSigmoid(float x)
{
  return 1 / (1 + exp(-x));
>>>>>>> simple-nn
}

__global__ void forwardPass(int nFeatures, int batchSize, int nHiddenLayer, int nOutput,
                            float *input, float *weights, float *weightsOutput, float *activationL1, float *result)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < batchSize)
  {

<<<<<<< HEAD
}

=======
    // Compute the activations of the hidden layer (Layer 1)
    for (int i = 0; i < nHiddenLayer; i++)
    {
      float hiddenSum = 0.0f;

      // Perform matrix multiplication between transposed input and weights
      for (int j = 0; j < nFeatures; j++)
      {
        hiddenSum += input[j * batchSize + tid] * weights[i * nFeatures + j];
      }

      // Store the activation of the hidden layer
      activationL1[tid * nHiddenLayer + i] = localSigmoid(hiddenSum);
    }

    // Compute the output layer activations
    for (int c = 0; c < nOutput; c++)
    {
      float sum = 0.0f;

      for (int i = 0; i < nHiddenLayer; i++)
      {
        sum += activationL1[tid * nHiddenLayer + i] * weightsOutput[i * nOutput + c];
      }

      // Apply activation function (e.g., sigmoid, ReLU, etc.) to the sum
      result[tid * nOutput + c] = exp(sum);
    }
    // Normalize the result to obtain probabilities using softmax
    float totalSum = 0.0f;
    for (int c = 0; c < nOutput; c++)
    {
      totalSum += result[tid * nOutput + c];
    }

    for (int c = 0; c < nOutput; c++)
    {
      result[tid * nOutput + c] /= totalSum;
    }
  }
}

__device__ void softmax(int nOutput, int batchSize, float *input)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < batchSize)
  {
    float maxVal = input[tid * nOutput];
    for (int i = 1; i < nOutput; i++)
    {
      maxVal = max(maxVal, input[tid * nOutput + i]);
    }

    float sum = 0.0f;
    for (int i = 0; i < nOutput; i++)
    {
      input[tid * nOutput + i] = exp(input[tid * nOutput + i] - maxVal);
      sum += input[tid * nOutput + i];
    }

    for (int i = 0; i < nOutput; i++)
    {
      input[tid * nOutput + i] /= sum;
    }
  }
}

__global__ void globalSoftmax(int nOutput, int batchSize, float *input)
{
  softmax(nOutput, batchSize, input);
}

__global__ void optimizedForwardPass(int nFeatures, int batchSize, int nHiddenLayer, int nOutput,
                                     float *input,
                                     float *weights, float * Z1, float *activationL1,
                                     float * weightsOutput, float *Z2, float *result)
{
  int bx = blockIdx.x;
  int tid = threadIdx.x;

  extern __shared__ float activationL1_s[];

  for (int i = tid; i < nHiddenLayer; i += blockDim.x)
  {
    float hiddenSum = 0.0f;
    for (int j = 0; j < nFeatures; j++)
    {
      hiddenSum += input[j * batchSize + bx] * weights[i * nFeatures + j];
    }
    Z1[bx * nHiddenLayer + i] = hiddenSum;
    activationL1_s[i] = localReLU(hiddenSum);
  }
  __syncthreads();

  for (int c = tid; c < nOutput; c += blockDim.x)
  {
    float sum = 0.0f;

    for (int i = 0; i < nHiddenLayer; i++)
    {
      sum += activationL1_s[i] * weightsOutput[i * nOutput + c];
    }
    Z2[bx * nOutput + c] = sum;
    result[bx * nOutput + c] = exp(sum);
  }
  // __syncthreads();
  float totalSum = 0.0f;
  for (int c = 0; c < nOutput; ++c)
  {
    totalSum += result[bx * nOutput + c];
  }

  // __syncthreads();
  for (int c = tid; c < nOutput; c += blockDim.x)
  {
    result[bx * nOutput + c] /= totalSum;
  }
  for (int i = tid; i < nHiddenLayer; i += blockDim.x)
  {
    activationL1[bx * nHiddenLayer + i] = activationL1_s[i];
  }
}

__global__ void categoricalCrossEntropy(int nOutput, int batchSize, float *groundTruth, float *predictions, float *loss)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < batchSize)
  {
    float example_loss = 0.0f;

    for (int c = 0; c < nOutput; c++)
    {
      example_loss -= groundTruth[tid * nOutput + c] * log(predictions[tid * nOutput + c]);
    }

    loss[tid] = example_loss;
  }
}

__global__ void backpropagation(int nFeatures, int batchSize, int nHiddenLayer, int nOutput,
                                     float *Z1, float *activationL1,
                                     float *Z2, float *result, float * weightsOutput,
                                     float *input, float *labels,
                                     float* dZ2, float* dW2, float* dZ1, float* dW1) {
  int bx = blockIdx.x;
  int tid = threadIdx.x;
  for (int i = tid; i < nOutput; i += blockDim.x)
  {
    dZ2[bx * nOutput + i] = result[bx * nOutput + i] - labels[bx * nOutput + i];
  }
  // dW2 = 1/m + dZ2 * activationL1.transpose()
  for (int i = tid; i < nHiddenLayer; i += blockDim.x)
  {
    float sum = 0.0f;
    for (int c = 0; c < nOutput; c++)
    {
      sum += dZ2[bx * nOutput + c] * activationL1[c*nHiddenLayer + i];
    }
    dW2[bx * nHiddenLayer + i] = sum / batchSize;
  }

  // dZ1 = weightsOutput.transpose() * dZ2 * localReLU'(Z1)
  for (int i = tid; i < nHiddenLayer; i += blockDim.x)
  {
    float sum = 0.0f;
    for (int c = 0; c < nOutput; c++)
    {
      sum += weightsOutput[c * nOutput + i] * dZ2[bx * nOutput + c];
    }
    dZ1[bx * nHiddenLayer + i] = sum * localReLUPrime(Z1[bx * nHiddenLayer + i]);
  }

  //dW1 = 1/m * dZ1 * input.transpose()
  for (int i = tid; i < nFeatures; i += blockDim.x)
  {
    float sum = 0.0f;
    for (int c = 0; c < nHiddenLayer; c++)
    {
      sum += dZ1[bx * nHiddenLayer + c] * input[c * nFeatures + i];
    }
    dW1[bx * nFeatures + i] = sum / nOutput;
  }
}

__device__ float localReLU(float x)
{
  return max(0.0f, x);
}

__device__ float localReLUPrime(float x)
{
  return x > 0.0f ? 1.0f : 0.0f;
}
>>>>>>> simple-nn
