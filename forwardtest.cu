#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "nnfunctions.h"
#include "utils.h"

// Define the necessary CUDA headers and functions here
void seqForwardPass(int nFeatures, int batchSize, int nHiddenLayer, int nOutput, float *input, float *weights, float *weightsOutput, float *activationL1, float *result);

float computeBatchCategoricalCrossEntropy(int nOutput, int batchSize, float *target, float *predicted);

int main()
{
    // define pointers to data
    float *d_input, *d_weights, *d_weightsOutput, *d_activation, *d_result, *d_temp;
    float *h_input, *h_weights, *h_weightsOutput, *h_activation, *h_result;
    hipEvent_t E0, E1, E2, E3;

    float *h_labels, *h_loss;
    float *d_labels, *d_loss;
    // srand(87);
    int nFeatures = 28 * 28;
    int batchSize = 64;
    int nOutput = 15;
    int nHiddenLayer = 1024;

    float totalTime;
    float seqTime;

    // Allocate host memory for the input, layers, and result arrays
    h_input = (float *)malloc(sizeof(float) * nFeatures * batchSize);
    h_weights = (float *)malloc(sizeof(float) * nFeatures * nHiddenLayer);
    h_weightsOutput = (float *)malloc(sizeof(float) * nHiddenLayer * nOutput);
    h_activation = (float *)malloc(sizeof(float) * nHiddenLayer * batchSize);
    h_result = (float *)malloc(sizeof(float) * nOutput * batchSize);
    h_labels = (float *)malloc(sizeof(float) * nOutput * batchSize);
    h_loss = (float *)malloc(sizeof(float) * batchSize);

    // Allocate device memory for the input, layers, and result arrays
    hipMalloc((void **)&d_input, sizeof(float) * nFeatures * batchSize);
    hipMalloc((void **)&d_weights, sizeof(float) * nFeatures * nHiddenLayer);
    hipMalloc((void **)&d_weightsOutput, sizeof(float) * nHiddenLayer * nOutput);
    hipMalloc((void **)&d_activation, sizeof(float) * nHiddenLayer * batchSize);
    hipMalloc((void **)&d_result, sizeof(float) * nOutput * batchSize);
    hipMalloc((void **)&d_temp, sizeof(float) * nFeatures * batchSize);
    hipMalloc((void **)&d_labels, sizeof(float) * nOutput * batchSize);
    hipMalloc((void **)&d_loss, sizeof(float) * batchSize);

    // Initialize the neural network weights with random values
    for (int i = 0; i < nFeatures * nHiddenLayer; i++)
    {

        h_weights[i] = -1.0f + 2.0f * rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < nHiddenLayer * nOutput; i++)
    {
        h_weightsOutput[i] = -1.0f + 2.0f * rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < nFeatures * batchSize; i++)
    {

        h_input[i] = -1.0f + 2.0f * rand() / (float)RAND_MAX;
    }

    for (int i = 0; i < nOutput * batchSize; i++)
    {
        if (i % nOutput == 0)
        {
            h_labels[i] = 1.0f;
        }
        else
        {
            h_labels[i] = 0.0f;
        }
    }
    hipEventCreate(&E0);
    hipEventCreate(&E1);
    hipEventCreate(&E2);
    hipEventCreate(&E3);

    // Copy the input and layer data to the device
    hipMemcpy(d_input, h_input, sizeof(float) * nFeatures * batchSize, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, h_weights, sizeof(float) * nFeatures * nHiddenLayer, hipMemcpyHostToDevice);
    hipMemcpy(d_weightsOutput, h_weightsOutput, sizeof(float) * nHiddenLayer * nOutput, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, h_labels, sizeof(float) * nOutput * batchSize, hipMemcpyHostToDevice);

    int nThreads = 32;
   int nBlocksN = (nHiddenLayer+nThreads-1)/nThreads; 
  int nBlocksM = (batchSize+nThreads-1)/nThreads; 


  dim3 grid(nBlocksM, nBlocksN, 1);
  dim3 block(nThreads, nThreads, 1);

    // Define the grid and block sizes for the CUDA kernel launch
    // dim3 grid(32, 32, 1);
    // dim3 block(32, 32, 1);

    // Call the forwardPass CUDA kernel
    
    hipEventRecord(E0, 0);
    hipEventSynchronize(E0);
    size_t sze = nHiddenLayer*sizeof(float);
    // forwardPass<<<batchSize, 512>>>(nFeatures, batchSize, nHiddenLayer, nOutput, d_input, d_weights, d_weightsOutput, d_activation, d_result);
    optimizedForwardPass<<<batchSize, 1024, sze>>>(nFeatures, batchSize, nHiddenLayer, nOutput, d_input, d_weights, d_weightsOutput, d_activation, d_result);
    hipError_t error = hipGetLastError();
    
    if (error != hipSuccess) {
        printf("CUDA error occurred: %s\n",
            hipGetErrorString(error));
    }


    hipEventRecord(E1, 0);
    hipEventSynchronize(E1);
    categoricalCrossEntropy<<<batchSize, nOutput>>>(nOutput, batchSize, d_labels, d_result, d_loss);

    hipMemcpy(h_loss, d_loss, sizeof(float) * batchSize, hipMemcpyDeviceToHost);


    // hipEventRecord(E0, 0);
    // hipEventSynchronize(E0);
    // // transpose<<<32,32>>>(batchSize, nFeatures, d_temp, d_input);

    // matMult<<<grid,block>>>(nHiddenLayer, batchSize, nFeatures, d_weights, d_input, d_activation);

    // // sigmoid<<<32,32>>>(nHiddenLayer*batchSize, d_activation, d_activation);

    // // matMult<<<grid,block>>>(nOutput, batchSize, nHiddenLayer, d_weightsOutput, d_activation, d_result);



    // // globalSoftmax<<<32,32>>>(nOutput,batchSize,d_result);
    // // hipMemcpy(h_result, d_result, sizeof(float) * nOutput * batchSize, hipMemcpyDeviceToHost);
    // hipEventRecord(E1, 0);
    // hipEventSynchronize(E1);
    
    
    hipEventElapsedTime(&totalTime, E0, E1);
    int numMatrixMult1Ops = batchSize * nFeatures * nHiddenLayer; // input x weights
    int numMatrixMult2Ops = batchSize * nHiddenLayer * nOutput;   // activationL1 x weightsOutput

    // Estimate the floating-point operations for the additions
    // Each matrix multiplication involves (nFeatures - 1) additions

    int numAdditionOps1 = (batchSize * nFeatures * (nHiddenLayer - 1));
    int numAdditionOps2 = (batchSize * nHiddenLayer * (nOutput - 1));

    // Total floating-point operations
    int totalFloatingPointOps = numMatrixMult1Ops + numMatrixMult2Ops + numAdditionOps1 + numAdditionOps2;

    // Copy the result data back to the host
    // hipMemcpy(h_result, d_result, sizeof(float) * nOutput * batchSize, hipMemcpyDeviceToHost);

    //     // Print the results
    //     for(int i = 0; i < 1; i++){
    //     float sum = 0;
    //     for(int j = 0; j < nOutput; j++){
    //         printf("batch %d, output %d: %f\n", i, j, h_result[i * nOutput + j]);
    //         sum+=h_result[i * nOutput + j];
    //     }
    //     printf("total: %f\n", sum);
    // }

    // forwardPass<<<32, 32>>>(nFeatures, batchSize, nHiddenLayer, nOutput, d_input, d_weights, d_weightsOutput, d_activation, d_result);
    hipMemcpy(h_result, d_result, sizeof(float) * nOutput * batchSize, hipMemcpyDeviceToHost);
    float *h_temp = (float *)malloc(sizeof(float) * nOutput * batchSize);
    hipEventRecord(E2, 0);
    hipEventSynchronize(E2);
    seqForwardPass(nFeatures, batchSize, nHiddenLayer, nOutput, h_input, h_weights, h_weightsOutput, h_activation, h_temp);
    hipEventRecord(E3, 0);
    hipEventSynchronize(E3);
    hipEventElapsedTime(&seqTime, E2, E3);

    int count = 0;

    for (int i = 0; i < batchSize; i++)
    {
        float sum1 = 0;
        float sum2 = 0;

        for (int j = 0; j < nOutput; j++)
        {
            if (abs(h_result[i * nOutput + j] - h_temp[i * nOutput + j]) > 0.0001)
            {
                printf("batch %d, output %d: %f, %f\n", i, j, h_result[i * nOutput + j], h_temp[i * nOutput + j]);
                ++count;
            }

            sum1 += h_result[i * nOutput + j];
            sum2 += h_temp[i * nOutput + j];
        }
    }
    printf("count: %d\n", count);
    float ls = computeBatchCategoricalCrossEntropy(nOutput, batchSize, h_labels, h_temp);
    float ls_d = 0.0f;
    for (int i = 0; i < batchSize; i++)
    {
        ls_d += h_loss[i];
    }
    printf("loss cuda: %f\n", ls_d / batchSize);
    printf("loss seq: %f\n", ls);

    printf("Total time: %4.6f milseg\n", totalTime);
    printf("Sequential time: %4.6f milseg\n", seqTime);
    printf("Total floating-point operations: %d\n", totalFloatingPointOps);
    printf("GFLOPs: %4.6f\n", totalFloatingPointOps / (totalTime * 1000000.0));
    printf("GFLOPs SEQ: %4.6f\n", totalFloatingPointOps / (seqTime * 1000000.0));
    printf("Speedup: %4.6f\n", seqTime / totalTime);

    // Free the device memory
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_weightsOutput);
    hipFree(d_activation);
    hipFree(d_result);
    hipFree(d_temp);

    // Free the host memory
    free(h_input);
    free(h_weights);
    free(h_weightsOutput);
    free(h_activation);
    free(h_result);
    free(h_temp);
}

void seqForwardPass(int nFeatures, int batchSize, int nHiddenLayer, int nOutput, float *input, float *weights, float *weightsOutput, float *activationL1, float *result)
{
    for (int tid = 0; tid < batchSize; tid++)
    {
        // Compute the activations of the hidden layer (Layer 1)
        for (int i = 0; i < nHiddenLayer; i++)
        {
            float hiddenSum = 0.0f;

            // Perform matrix multiplication between transposed input and weights
            for (int j = 0; j < nFeatures; j++)
            {
                hiddenSum += input[j * batchSize + tid] * weights[i * nFeatures + j];
            }

            // Store the activation of the hidden layer
            activationL1[tid * nHiddenLayer + i] = 1 / (1 + exp(-hiddenSum));
        }

        // Compute the output layer activations
        for (int c = 0; c < nOutput; c++)
        {
            float sum = 0.0f;

            for (int i = 0; i < nHiddenLayer; i++)
            {
                sum += activationL1[tid * nHiddenLayer + i] * weightsOutput[i * nOutput + c];
            }

            // Apply activation function (e.g., sigmoid, ReLU, etc.) to the sum
            result[tid * nOutput + c] = exp(sum);
        }

        // Normalize the result to obtain probabilities using softmax
        float totalSum = 0.0f;
        for (int c = 0; c < nOutput; c++)
        {
            totalSum += result[tid * nOutput + c];
        }

        for (int c = 0; c < nOutput; c++)
        {
            result[tid * nOutput + c] /= totalSum;
        }
    }
}

float computeBatchCategoricalCrossEntropy(int nOutput, int batchSize, float *target, float *predicted)
{
    float totalLoss = 0.0f;

    for (int i = 0; i < batchSize; i++)
    {
        float loss = 0.0f;

        for (int c = 0; c < nOutput; c++)
        {
            loss -= target[i * nOutput + c] * log(predicted[i * nOutput + c]);
        }

        totalLoss += loss;
    }

    return totalLoss / batchSize;
}
